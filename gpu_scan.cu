#include "hip/hip_runtime.h"
#include "gpu_scan.cuh"
#include <hip/hip_runtime.h>
#include <vector>
#include <array>
#include <iostream>

extern __device__ void sha256(const uint8_t* data, size_t len, uint8_t* out32);
extern __device__ void ripemd160(const uint8_t* data, size_t len, uint8_t* out20);

__device__ void generate_fake_pubkey(uint64_t privkey, uint8_t* out33) {
    out33[0] = 0x02; // Compressed
    for (int i = 1; i < 33; i++) out33[i] = (privkey >> ((i-1)*2)) & 0xFF;
}

__global__ void scan_kernel(uint64_t start, uint64_t total, const uint8_t* d_targets, size_t num_targets,
                            uint64_t* d_matches, uint8_t* d_hashes, int* d_count) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    uint64_t key = start + idx;

    uint8_t pubkey[33], sha[32], h160[20];
    generate_fake_pubkey(key, pubkey);
    sha256(pubkey, 33, sha);
    ripemd160(sha, 32, h160);

    for (int t = 0; t < num_targets; t++) {
        bool match = true;
        for (int j = 0; j < 20; j++) {
            if (h160[j] != d_targets[t * 20 + j]) {
                match = false;
                break;
            }
        }

        if (match) {
            int pos = atomicAdd(d_count, 1);
            d_matches[pos] = key;
            for (int j = 0; j < 20; j++)
                d_hashes[pos * 20 + j] = h160[j];
        }
    }
}

std::vector<std::pair<uint64_t, std::array<uint8_t, 20>>> scan_range_on_gpu_with_output(
    uint64_t start, uint64_t end, const std::vector<std::array<uint8_t, 20>>& targets
) {
    uint64_t total = end - start;
    std::vector<std::pair<uint64_t, std::array<uint8_t, 20>>> results;

    std::vector<uint8_t> flat_targets;
    for (auto& t : targets) flat_targets.insert(flat_targets.end(), t.begin(), t.end());

    uint8_t* d_targets;
    uint64_t* d_matches;
    uint8_t* d_hashes;
    int* d_count;
    hipMalloc(&d_targets, flat_targets.size());
    hipMalloc(&d_matches, total * sizeof(uint64_t));
    hipMalloc(&d_hashes, total * 20);
    hipMalloc(&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));
    hipMemcpy(d_targets, flat_targets.data(), flat_targets.size(), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    scan_kernel<<<blocks, threads>>>(start, total, d_targets, targets.size(), d_matches, d_hashes, d_count);
    hipDeviceSynchronize();

    int h_count = 0;
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    std::vector<uint64_t> h_keys(h_count);
    std::vector<uint8_t> h_h160(h_count * 20);
    hipMemcpy(h_keys.data(), d_matches, h_count * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_h160.data(), d_hashes, h_count * 20, hipMemcpyDeviceToHost);

    for (int i = 0; i < h_count; i++) {
    std::array<uint8_t, 20> hash;
    for (int j = 0; j < 20; ++j) {
        hash[j] = h_h160[i * 20 + j];
    }
    results.emplace_back(h_keys[i], hash);
    }

    hipFree(d_targets);
    hipFree(d_matches);
    hipFree(d_hashes);
    hipFree(d_count);

    return results;
}
